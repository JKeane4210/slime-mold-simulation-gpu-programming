#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <stdlib.h>
#include <time.h>
#include <errno.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
// CUDA helper functions
// #include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#include <GL/freeglut.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <hip/hip_runtime_api.h>
#include <helper_gl.h>

// Utilities and timing functions
// #include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

#define BLOCK_SIZE 32
#define BLOCK_SIZE_PARTICLE 512
#define DEBUG_LEVEL 0
#define HARD_FAIL false

#define RA M_PI / 4 // = rotation angle
#define SA M_PI / 4 // = sensor angle
#define SO 9 // pixel(s) = sensor offset (original = 9)
#define SW 1 // pixel(s) = sensor width
#define SS 1 // pixel(s) = step size
#define depT 20 // how much chemoattractant is deposited (original = 5)
#define decayT 0.5 // decay rate of chemoattractant
#define ENV_WIDTH 400
#define ENV_HEIGHT 300

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width  = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width    = 256;
const unsigned int mesh_height   = 256;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

bool runTest(int argc, char **argv, char *ref_file);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource);
void runAutoTest(int devID, char **argv, char *ref_file);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);

const char *sSDKsample = "simpleGL (VBO)";


// float env[ENV_HEIGHT][ENV_WIDTH];
// bool occupied[ENV_HEIGHT][ENV_WIDTH];

struct SlimeParticle {
    float x;
    float y;
    float orientation;
    hiprandState_t rng;
};

#define MASK_WIDTH 3    
__constant__ float K[MASK_WIDTH][MASK_WIDTH];

//handle error macro
static void HandleError(hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line );
    }
}
    
#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))

// LEARNING POINT - If I am trying to initialize randomly for each point and synchronize,
// it seems like this will be a very difficult task. I can think of it however as every pixel
// having a probability of being taken and then I might get closer to what I am looking for

__global__ 
void init_particle_kernel(SlimeParticle * particles, int n, int * occupied, int w, int h) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        hiprandState_t state;
        hiprand_init(i, 0, 0, &state); // Initialize the generator
        int rx = ((int)ceilf(hiprand_uniform(&state) * w)) - 1;
        int ry = ((int)ceilf(hiprand_uniform(&state) * h)) - 1;
        float r_theta = hiprand_uniform(&state) * 2 * M_PI;
        while (atomicAdd(&(occupied[ry * w + rx]), 1) != 0) {
            rx = ((int)ceilf(hiprand_uniform(&state) * w)) - 1;
            ry = ((int)ceilf(hiprand_uniform(&state) * h)) - 1;
        }
        particles[i].x = (float)rx;
        particles[i].y = (float)ry;
        particles[i].orientation = r_theta;
        particles[i].rng = state;
    }
}

void report_kernel_benchmark(const char * kernel_name, hipEvent_t start, hipEvent_t stop) {
    float benchmark_ms = 0;
    hipEventSynchronize(stop); //wait for the stop event, if it isn’t done
    hipEventElapsedTime(&benchmark_ms, start, stop); //get the elapsed time
    printf("- Kernel: %s, Benchmark(ms): %f\n", kernel_name, benchmark_ms);
}

// elapsed time in milliseconds
float cpu_time(timespec* start, timespec* end){
    return ((1e9*end->tv_sec + end->tv_nsec) - (1e9*start->tv_sec + start->tv_nsec))/1e6;
}

__device__
float sample_chemoattractant(SlimeParticle* p, float * env, int w, int h, float rotation_offset, float sensor_offset) {
    float angle = p->orientation + rotation_offset;
    if (angle < 0) angle += 2 * M_PI;
    if (angle > 2 * M_PI) angle -= 2 * M_PI;
    int s_x = (int)round(p->x + sensor_offset * cos(angle));
    int s_y = (int)round(p->y + sensor_offset + sin(angle));
    if (s_y >= 0 && s_y < ENV_HEIGHT && s_x >= 0 && s_x < ENV_WIDTH)
        return env[s_y * w + s_x];
    else
        p->orientation += hiprand_uniform(&(p->rng)) * M_PI_2 + M_PI_4; // keep it in bounds
        if (p->orientation > 2 * M_PI) p->orientation -= 2 * M_PI;
        return 0;
}

__global__
void sensor_stage_kernel(SlimeParticle * particles, int n, float * env, int w, int h, float sensor_angle, float rotation_angle) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        SlimeParticle * p = &particles[i];
        float F = sample_chemoattractant(p, env, w, h, 0, SO);
        float FR = sample_chemoattractant(p, env, w, h, -sensor_angle, SO);
        float FL = sample_chemoattractant(p, env, w, h, sensor_angle, SO);
        if ((F > FL) && (F > FR)) return;
        else if ((F < FL) && (F < FR)) {
            int random_sign = (int)(ceilf(hiprand_uniform(&(p->rng)) * 2)) - 1;
            p->orientation += (random_sign ? 1 : -1) * rotation_angle;
        } else if (FL < FR) {
            p->orientation -= rotation_angle;
        } else if (FR < FL) {
            p->orientation += rotation_angle;
        }
        if (p->orientation < 0) p->orientation += 2 * M_PI;
        if (p->orientation > 2 * M_PI) p->orientation -= 2 * M_PI;
    }
}

__global__
void motor_stage_kernel(SlimeParticle * particles, int n, float * env, int * occupied, int w, int h, float sensor_angle, float rotation_angle) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        SlimeParticle * p = &particles[i];
        float n_x = p->x + cos(p->orientation) * SS;
        float n_y = p->y + sin(p->orientation) * SS;
        int p_x_i = (int)round(p->x);
        int p_y_i = (int)round(p->y);
        int n_x_i = (int)round(n_x);
        int n_y_i = (int)round(n_y);
        if (atomicAdd(&(occupied[n_y_i * w + n_x_i]), 1) == 0) { // not occupied
            atomicExch(&(occupied[p_y_i * w + p_x_i]), 0); // clear previous location
            p->x = n_x;
            p->y = n_y;
            if (p_y_i < h && p_y_i >= 0 && p_x_i < w && p_x_i >= 0)
                atomicAdd(&(env[p_y_i * w + p_x_i]), depT); // deposit trail in new location
        } else {
            p->orientation += hiprand_uniform(&(p->rng)) * M_PI_2 - M_PI_4; // choose random new orientation
            if (p->orientation < 0) p->orientation += 2 * M_PI;
            if (p->orientation > 2 * M_PI) p->orientation -= 2 * M_PI;
        }
    }
}

__global__
void decay_chemoattractant_kernel(float * env, int w, int h) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    if (col < w && col >= 0 && row < h && row >= 0) {
        env[row * w + col] = max(env[row * w + col] - decayT, 0.0);
    }
}

void gray_scale_image_to_file(const char *cpu_output_file, float * env) {
    // printf("Filename: %s\n", cpu_output_file);
    
    FILE *output_file_handle = fopen(cpu_output_file, "w");
    if (output_file_handle == NULL) {
        // Print a descriptive error message
        perror("Error opening file");
        
        // Alternatively, use strerror to get the error message
        printf("fopen failed: %s\n", strerror(errno));
        return;
    }
    fprintf(output_file_handle, "%s\n#\n%d %d\n%d\n", "P5", ENV_WIDTH, ENV_HEIGHT, 255);
    for (int i = 0; i < ENV_HEIGHT; ++i)
    {
        for (int j = 0; j < ENV_WIDTH; ++j)
        {
            fputc((int)(min(255.0, env[i * ENV_WIDTH + j])), output_file_handle); // TODO
        }
    }
    fflush(output_file_handle);
    fclose(output_file_handle);
}

// __global__
// void test_kernel() {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     int test = 0;
//     printf("r1: %d\n", atomicAdd(&test, 1));
//     printf("r2: %d\n", atomicAdd(&test, 1));
//     printf("%d\n", test);
// }

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();

    g_fAnim += 0.01f;

    sdkStopTimer(&timer);
    computeFPS();
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            #if defined(__APPLE__) || defined(MACOSX)
                exit(EXIT_SUCCESS);
            #else
                glutDestroyWindow(glutGetWindow());
                return;
            #endif
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);


    // First initialize OpenGL context, so we can properly set the GL for CUDA.
    // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
    if (false == initGL(&argc, argv))
    {
        return false;
    }

    // register callbacks
    glutDisplayFunc(display); // done
    glutKeyboardFunc(keyboard); // done
    glutMouseFunc(mouse); // done
    glutMotionFunc(motion); // done
    glutCloseFunc(cleanup); // done

    // create VBO
    createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

    // run the cuda part
    runCuda(&cuda_vbo_resource);

    // start rendering mainloop
    glutMainLoop();

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    // execute the kernel
    //    dim3 block(8, 8, 1);
    //    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

    // launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    char *ref_file = NULL;

#if defined(__linux__)
    setenv ("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", sSDKsample);

    printf("\n");

    runTest(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}



// int main(int argc, char* argv[]) {
//     int * occupied_d;
//     HANDLE_ERROR(hipMalloc((void **)&occupied_d, ENV_WIDTH * ENV_HEIGHT * sizeof(int)));
//     HANDLE_ERROR(hipMemset(occupied_d, 0, ENV_WIDTH * ENV_HEIGHT * sizeof(int)));

//     float * env_d;
//     float * env_h = (float *)malloc(ENV_WIDTH * ENV_HEIGHT * sizeof(float));
//     HANDLE_ERROR(hipMalloc((void **)&env_d, ENV_WIDTH * ENV_HEIGHT * sizeof(float)));
//     HANDLE_ERROR(hipMemset(env_d, 0, ENV_WIDTH * ENV_HEIGHT * sizeof(float)));

//     const int N_PARTICLES = 1600;
//     SlimeParticle* particles_d;
//     HANDLE_ERROR(hipMalloc((void **)&particles_d, N_PARTICLES * sizeof(SlimeParticle)));
//     printf("Dims: %d %d\n", (N_PARTICLES - 1) / BLOCK_SIZE_PARTICLE + 1, BLOCK_SIZE_PARTICLE);
//     init_particle_kernel<<<(N_PARTICLES - 1) / BLOCK_SIZE_PARTICLE + 1, BLOCK_SIZE_PARTICLE>>>(particles_d, N_PARTICLES, occupied_d, ENV_WIDTH, ENV_HEIGHT);
//     HANDLE_ERROR( hipPeekAtLastError() );
//     HANDLE_ERROR( hipDeviceSynchronize() );

//     // debug because dgb is being weird
//     // SlimeParticle* particles_h = (SlimeParticle *)malloc(N_PARTICLES * sizeof(SlimeParticle));
//     // HANDLE_ERROR(hipMemcpy(particles_h, particles_d, N_PARTICLES * sizeof(SlimeParticle), hipMemcpyDeviceToHost));
//     // printf("Size: %lu\n", N_PARTICLES * sizeof(SlimeParticle));
//     // for (int i = 0; i < N_PARTICLES; ++i) {
//     //     printf("Particle: %.2f, %.2f @ %.3f\n", particles_h[i].x, particles_h[i].y, particles_h[i].orientation);
//     // }

//     dim3 dg((ENV_WIDTH - 1) / BLOCK_SIZE + 1, (ENV_WIDTH - 1) / BLOCK_SIZE + 1, 1);
//     dim3 db(32, 32, 1);
//     char buffer[300];
//     const int N_STEPS = 1000;
//     for (int i = 0; i < N_STEPS; ++i) {
//         // update step
//         sensor_stage_kernel<<<(N_PARTICLES - 1) / BLOCK_SIZE_PARTICLE + 1, BLOCK_SIZE_PARTICLE>>>(particles_d, N_PARTICLES, env_d, ENV_WIDTH, ENV_HEIGHT, SA, RA);
//         motor_stage_kernel<<<(N_PARTICLES - 1) / BLOCK_SIZE_PARTICLE + 1, BLOCK_SIZE_PARTICLE>>>(particles_d, N_PARTICLES, env_d, occupied_d, ENV_WIDTH, ENV_HEIGHT, SA, RA);
//         decay_chemoattractant_kernel<<<dg, db>>>(env_d, ENV_WIDTH, ENV_HEIGHT);
        
//         // save image to file
//         hipMemcpy(env_h, env_d, ENV_WIDTH * ENV_HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
//         sprintf(buffer, "./Desktop/Projects/slime-mold-simulation-gpu-programming/frames/frame_%d.ppm", i);
//         gray_scale_image_to_file(buffer, env_h);
//     }

//     HANDLE_ERROR(hipFree(particles_d));
//     HANDLE_ERROR(hipFree(env_d));
//     HANDLE_ERROR(hipFree(occupied_d));

//     printf("Done!\n");
    
//     // init environment
//     // for (int i = 0; i < ENV_HEIGHT; ++i) {
//     //     for (int j = 0; j < ENV_WIDTH; ++j) {
//     //         env[i][j] = 0.0;
//     //         occupied[i][j] = false;
//     //     }
//     // }
    
//     // random init particles in center
//     // for (int i = 0; i < 10; ++i) {
//     //     for (int j = 0; j < 10; ++j) {
//     //         float random_orientation = ((float)rand() / (float)RAND_MAX) * 2 * M_PI;

//     //         particles[i * 10 + j] = { (float)(i + ENV_WIDTH / 2 - 5),  (float)(j + ENV_HEIGHT / 2 - 5), random_orientation};
//     //         printf("Particle: %.3f %.3f @ %.3f\n", particles[i * 10 + j].x, particles[i * 10 + j].y, particles[i * 10 + j].orientation);
//     //         occupied[i + ENV_HEIGHT / 2 - 5][j + ENV_WIDTH / 2 - 5] = true;
//     //         env[i + ENV_HEIGHT / 2 - 5][j + ENV_WIDTH / 2 - 5] = depT; // initial deposit
//     //     }
//     // }

//     // circle init
//     // const float RADIUS_MAX = 45;
//     // for (int i = 0; i < N_PARTICLES; ++i) {
//     //     float r_i = ((float)rand() / (float)RAND_MAX) * RADIUS_MAX;
//     //     float theta_i = ((float)rand() / (float)RAND_MAX) * 2 * M_PI;
//     //     int y_i = ENV_HEIGHT / 2 + (int)(r_i * sin(theta_i));
//     //     int x_i = ENV_WIDTH / 2 + (int)(r_i * cos(theta_i));
//     //     while (occupied[y_i][x_i]) {
//     //         r_i = ((float)rand() / (float)RAND_MAX) * RADIUS_MAX;
//     //         theta_i = ((float)rand() / (float)RAND_MAX) * 2 * M_PI;
//     //         y_i = ENV_HEIGHT / 2 + (int)(r_i * sin(theta_i));
//     //         x_i = ENV_WIDTH / 2 + (int)(r_i * cos(theta_i));
//     //     }
//     //     float random_orientation = ((float)rand() / (float)RAND_MAX) * 2 * M_PI;
//     //     particles[i] = { (float)x_i, (float)y_i, random_orientation};
//     //     printf("Particle: %.3f %.3f @ %.3f\n", particles[i].x, particles[i].y, particles[i].orientation);
//     //     occupied[y_i][x_i] = true;
//     //     env[y_i][x_i] = depT; // initial deposit
//     // }

//     // char buffer[300];
//     // // simulation loop
//     // for (int step = 0; step < 1000; ++step) {
//     //     for (int i = 0; i < N_PARTICLES; ++i) 
//     //         sensory_stage_cpu(&particles[i], SA, RA);
//     //     for (int i = 0; i < N_PARTICLES; ++i)
//     //         motor_stage_cpu(&particles[i]);
//     //     decay_chemoattractant();
        
//     //     sprintf(buffer, "./source/final-project/frames/frame_%d.ppm", step);
//     //     gray_scale_image_to_file(buffer);
//     // }

//     return 0;
// }
