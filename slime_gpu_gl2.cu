#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <stdlib.h>
#include <time.h>
#include <errno.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

// libraries from NVIDIA samples
#include "libs/helper_gl.h"
#include "libs/helper_timer.h"
#include "libs/helper_image.h"
#include "libs/helper_string.h"
#include "libs/hip/hip_runtime_api.h"

// OpenGL libraries
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>

#define BLOCK_SIZE 32
#define BLOCK_SIZE_PARTICLE 256
#define DEBUG_LEVEL 0
#define HARD_FAIL false

#define RA M_PI / 8 // = rotation angle
#define SA M_PI / 4 // = sensor angle
#define SO 9        // pixel(s) = sensor offset (original = 9)
#define SW 1        // pixel(s) = sensor width
#define SS 1        // pixel(s) = step size
#define depT 20     // how much chemoattractant is deposited (original = 5)
#define decayT 0.5  // decay rate of chemoattractant
#define ENV_WIDTH 1000
#define ENV_HEIGHT 1000
#define N_PARTICLES 100000
#define DISPLAY_WIDTH 400
#define DISPLAY_HEIGHT 400

#define REFRESH_DELAY 10 // ms

struct SlimeParticle
{
    float x;
    float y;
    float orientation;
    hiprandState_t rng;
};

#define MASK_WIDTH 3
__constant__ float K[MASK_WIDTH][MASK_WIDTH];

void display();

StopWatchInterface *timer;
StopWatchInterface *kernel_timer;

unsigned int *img_host;
unsigned int *img_device;
unsigned int *tmp_img_device;

SlimeParticle *particles_d;
int *occupied_d;
float *env_h;
float *env_d;

dim3 dg((ENV_WIDTH - 1) / BLOCK_SIZE + 1, (ENV_WIDTH - 1) / BLOCK_SIZE + 1, 1);
dim3 db(32, 32, 1);

unsigned int frameCount = 0;
unsigned int fpsCount = 0;
unsigned int fpsLimit = 8;
float avgFPS = 0.0f;

// what is a PBO? -> pixel buffer object
hipGraphicsResource *cuda_pbo_resource;
GLuint pbo;
GLuint texture_id;
GLuint shader;

// handle error macro
inline void HandleError(hipError_t err, const char *file, const int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void init_particle_kernel(SlimeParticle *particles, int n, int *occupied, int w, int h)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        hiprandState_t state;
        hiprand_init(i, 0, 0, &state); // Initialize the generator
        int rx = ((int)ceilf(hiprand_uniform(&state) * w)) - 1;
        int ry = ((int)ceilf(hiprand_uniform(&state) * h)) - 1;
        float r_theta = hiprand_uniform(&state) * 2 * M_PI;
        while (atomicAdd(&(occupied[ry * w + rx]), 1) != 0)
        {
            rx = ((int)ceilf(hiprand_uniform(&state) * w)) - 1;
            ry = ((int)ceilf(hiprand_uniform(&state) * h)) - 1;
        }
        particles[i].x = (float)rx;
        particles[i].y = (float)ry;
        particles[i].orientation = r_theta;
        particles[i].rng = state;
    }
}

void report_kernel_benchmark(const char *kernel_name, hipEvent_t start, hipEvent_t stop)
{
    float benchmark_ms = 0;
    hipEventSynchronize(stop);                       // wait for the stop event, if it isn’t done
    hipEventElapsedTime(&benchmark_ms, start, stop); // get the elapsed time
    printf("- Kernel: %s, Benchmark(ms): %f\n", kernel_name, benchmark_ms);
}

// elapsed time in milliseconds
float cpu_time(timespec *start, timespec *end)
{
    return ((1e9 * end->tv_sec + end->tv_nsec) - (1e9 * start->tv_sec + start->tv_nsec)) / 1e6;
}

__device__ float sample_chemoattractant(SlimeParticle *p, float *env, int w, int h, float rotation_offset, float sensor_offset)
{
    float angle = p->orientation + rotation_offset;
    if (angle < 0)
        angle += 2 * M_PI;
    if (angle > 2 * M_PI)
        angle -= 2 * M_PI;
    int s_x = (int)round(p->x + sensor_offset * cos(angle));
    int s_y = (int)round(p->y + sensor_offset + sin(angle));
    if (s_y >= 0 && s_y < ENV_HEIGHT && s_x >= 0 && s_x < ENV_WIDTH)
        return env[s_y * w + s_x];
    else
        p->orientation += hiprand_uniform(&(p->rng)) * M_PI_2 + M_PI_4; // keep it in bounds
    if (p->orientation > 2 * M_PI)
        p->orientation -= 2 * M_PI;
    return 0;
}

__global__ void sensor_stage_kernel(SlimeParticle *particles, int n, float *env, int w, int h, float sensor_angle, float rotation_angle)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        SlimeParticle *p = &particles[i];
        float F = sample_chemoattractant(p, env, w, h, 0, SO);
        float FR = sample_chemoattractant(p, env, w, h, -sensor_angle, SO);
        float FL = sample_chemoattractant(p, env, w, h, sensor_angle, SO);
        if ((F > FL) && (F > FR))
            return;
        else if ((F < FL) && (F < FR))
        {
            int random_sign = (int)(ceilf(hiprand_uniform(&(p->rng)) * 2)) - 1;
            p->orientation += (random_sign ? 1 : -1) * rotation_angle;
        }
        else if (FL < FR)
        {
            p->orientation -= rotation_angle;
        }
        else if (FR < FL)
        {
            p->orientation += rotation_angle;
        }
        if (p->orientation < 0)
            p->orientation += 2 * M_PI;
        if (p->orientation > 2 * M_PI)
            p->orientation -= 2 * M_PI;
    }
}

__global__ void motor_stage_kernel(SlimeParticle *particles, int n, float *env, int *occupied, int w, int h, float sensor_angle, float rotation_angle)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        SlimeParticle *p = &particles[i];
        float n_x = p->x + cos(p->orientation) * SS;
        float n_y = p->y + sin(p->orientation) * SS;
        int p_x_i = (int)round(p->x);
        int p_y_i = (int)round(p->y);
        int n_x_i = (int)round(n_x);
        int n_y_i = (int)round(n_y);
        if (atomicAdd(&(occupied[n_y_i * w + n_x_i]), 1) == 0)
        {                                                  // not occupied
            atomicExch(&(occupied[p_y_i * w + p_x_i]), 0); // clear previous location
            p->x = n_x;
            p->y = n_y;
            if (p_y_i < h && p_y_i >= 0 && p_x_i < w && p_x_i >= 0)
                atomicAdd(&(env[p_y_i * w + p_x_i]), depT); // deposit trail in new location
        }
        else
        {
            p->orientation += hiprand_uniform(&(p->rng)) * M_PI_2 - M_PI_4; // choose random new orientation
            if (p->orientation < 0)
                p->orientation += 2 * M_PI;
            if (p->orientation > 2 * M_PI)
                p->orientation -= 2 * M_PI;
        }
    }
}

__global__ void decay_chemoattractant_kernel(float *env, uint *result, int w, int h)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    if (col < w && col >= 0 && row < h && row >= 0)
    {
        float value = max(env[row * w + col] - decayT, 0.0);
        env[row * w + col] = value;
        value = min(value, 255.0);
        result[row * w + col] = (255u << 24) |
                                ((unsigned int)(value) << 16) |
                                ((unsigned int)(value) << 8) |
                                ((unsigned int)(value));
        // unsigned int test_val = value > 0 ? 0xffu : 0x00u;
        // result[row * w  + col] = (255u << 24) | (test_val << 16) | (test_val << 8) | (test_val);
    }
}

// Keyboard callback function for OpenGL (GLUT)
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
    case 27:
        glutDestroyWindow(glutGetWindow());
        return;
        break;

    default:
        break;
    }
}

// Resizing the window
void reshape(int x, int y)
{
    glViewport(0, 0, x, y);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0);
}

// Timer Event so we can refresh the display
void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
    }
}

void initGL(int *argc, char **argv)
{
    // initialize GLUT
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE); // only allow RGBA (can | GLUT_FLOAT to get other)
    glutInitWindowSize(DISPLAY_WIDTH, DISPLAY_HEIGHT);
    glutCreateWindow("CUDA Slime Mold Simulation");
    glutDisplayFunc(display);

    glutKeyboardFunc(keyboard);
    glutReshapeFunc(reshape);
    glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
}

void initCuda()
{
    HANDLE_ERROR(hipMalloc((void **)&img_device, ENV_WIDTH * ENV_HEIGHT * sizeof(unsigned int)));
    HANDLE_ERROR(hipMalloc((void **)&tmp_img_device, ENV_WIDTH * ENV_HEIGHT * sizeof(unsigned int)));

    HANDLE_ERROR(hipMalloc((void **)&occupied_d, ENV_WIDTH * ENV_HEIGHT * sizeof(int)));
    HANDLE_ERROR(hipMemset(occupied_d, 0, ENV_WIDTH * ENV_HEIGHT * sizeof(int)));

    // creating environment
    env_h = (float *)malloc(ENV_WIDTH * ENV_HEIGHT * sizeof(float));
    HANDLE_ERROR(hipMalloc((void **)&env_d, ENV_WIDTH * ENV_HEIGHT * sizeof(float)));
    HANDLE_ERROR(hipMemset(env_d, 0, ENV_WIDTH * ENV_HEIGHT * sizeof(float)));

    // creating array of particles
    HANDLE_ERROR(hipMalloc((void **)&particles_d, N_PARTICLES * sizeof(SlimeParticle)));
    printf("Dims: %d %d\n", (N_PARTICLES - 1) / BLOCK_SIZE_PARTICLE + 1, BLOCK_SIZE_PARTICLE);
    init_particle_kernel<<<(N_PARTICLES - 1) / BLOCK_SIZE_PARTICLE + 1, BLOCK_SIZE_PARTICLE>>>(particles_d, N_PARTICLES, occupied_d, ENV_WIDTH, ENV_HEIGHT);
    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    // creating timers
    sdkCreateTimer(&timer);
    sdkCreateTimer(&kernel_timer);
}

// Calculate the Frames per second and print in the title bar
void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.0f / (sdkGetAverageTimerValue(&timer) / 1000.0f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.0f);
        sdkResetTimer(&timer);
    }
}

void display()
{
    sdkStartTimer(&timer);

    // execute filter, writing results to pbo
    unsigned int *d_result;

    HANDLE_ERROR(hipGraphicsMapResources(1, &cuda_pbo_resource, 0));
    size_t num_bytes;
    HANDLE_ERROR(hipGraphicsResourceGetMappedPointer(
        (void **)&d_result, &num_bytes, cuda_pbo_resource)); // should come back saying ENV_WIDTH * ENV_HEIGHT * 4 bytes
    // printf("Bytes accessible: %zu\n", num_bytes);

    // update step
    sensor_stage_kernel<<<(N_PARTICLES - 1) / BLOCK_SIZE_PARTICLE + 1, BLOCK_SIZE_PARTICLE>>>(particles_d, N_PARTICLES, env_d, ENV_WIDTH, ENV_HEIGHT, SA, RA);
    // HANDLE_ERROR(hipPeekAtLastError());
    motor_stage_kernel<<<(N_PARTICLES - 1) / BLOCK_SIZE_PARTICLE + 1, BLOCK_SIZE_PARTICLE>>>(particles_d, N_PARTICLES, env_d, occupied_d, ENV_WIDTH, ENV_HEIGHT, SA, RA);
    // HANDLE_ERROR(hipPeekAtLastError());
    decay_chemoattractant_kernel<<<dg, db>>>(env_d, d_result, ENV_WIDTH, ENV_HEIGHT);
    // HANDLE_ERROR(hipPeekAtLastError());

    HANDLE_ERROR(hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0));
    HANDLE_ERROR(hipDeviceSynchronize());

    // if want to save a frame to .ppm
    // hipMemcpy((unsigned char *)h_result, (unsigned char *)d_result,
    //            ENV_WIDTH * ENV_HEIGHT * sizeof(unsigned int), hipMemcpyDeviceToHost);
    // sdkSavePPM4ub((const char *)"tmp.ppm", (unsigned char *)h_result, ENV_WIDTH, ENV_HEIGHT);

    // OpenGL display code path
    {
        glClear(GL_COLOR_BUFFER_BIT);

        // load texture from pbo
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
        glBindTexture(GL_TEXTURE_2D, texture_id);
        glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, ENV_WIDTH, ENV_HEIGHT, GL_RGBA,
                        GL_UNSIGNED_BYTE, 0);
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

        // fragment program is required to display floating point texture
        glBindProgramARB(GL_FRAGMENT_PROGRAM_ARB, shader);
        glEnable(GL_FRAGMENT_PROGRAM_ARB);
        glDisable(GL_DEPTH_TEST);

        glBegin(GL_QUADS);
        {
            glTexCoord2f(0.0f, 0.0f);
            glVertex2f(0.0f, 0.0f);
            glTexCoord2f(1.0f, 0.0f);
            glVertex2f(1.0f, 0.0f);
            glTexCoord2f(1.0f, 1.0f);
            glVertex2f(1.0f, 1.0f);
            glTexCoord2f(0.0f, 1.0f);
            glVertex2f(0.0f, 1.0f);
        }
        glEnd();
        glBindTexture(GL_TEXTURE_2D, 0);
        glDisable(GL_FRAGMENT_PROGRAM_ARB);
    }

    glutSwapBuffers();
    glutReportErrors();

    sdkStopTimer(&timer);

    computeFPS();
}

// shader for displaying floating-point texture
static const char *shader_code =
    "!!ARBfp1.0\n"
    "TEX result.color, fragment.texcoord, texture[0], 2D; \n"
    "END";

GLuint compileASMShader(GLenum program_type, const char *code)
{
    GLuint program_id;
    glGenProgramsARB(1, &program_id);
    glBindProgramARB(program_type, program_id);
    glProgramStringARB(program_type, GL_PROGRAM_FORMAT_ASCII_ARB,
                       (GLsizei)strlen(code), (GLubyte *)code);

    GLint error_pos;
    glGetIntegerv(GL_PROGRAM_ERROR_POSITION_ARB, &error_pos);

    if (error_pos != -1)
    {
        const GLubyte *error_string;
        error_string = glGetString(GL_PROGRAM_ERROR_STRING_ARB);
        printf("Program error at position: %d\n%s\n", (int)error_pos, error_string);
        return 0;
    }

    return program_id;
}

// This is where we create the OpenGL PBOs, FBOs, and texture resources
void initGLResources()
{
    // create pixel buffer object
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, ENV_WIDTH * ENV_HEIGHT * sizeof(GLubyte) * 4,
                 img_host, GL_STREAM_DRAW_ARB);

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    HANDLE_ERROR(hipGraphicsGLRegisterBuffer(
        &cuda_pbo_resource, pbo, cudaGraphicsMapFlagsWriteDiscard));

    // create texture for display
    glGenTextures(1, &texture_id);
    glBindTexture(GL_TEXTURE_2D, texture_id);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, ENV_WIDTH, ENV_HEIGHT, 0, GL_RGBA,
                 GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glBindTexture(GL_TEXTURE_2D, 0);

    // load shader program
    shader = compileASMShader(GL_FRAGMENT_PROGRAM_ARB, shader_code);
}

void cleanup()
{
    // deleting timers
    sdkDeleteTimer(&timer);
    sdkDeleteTimer(&kernel_timer);

    if (img_host)
    {
        free(img_host);
        img_host = NULL;
    }

    if (img_device)
    {
        hipFree(img_device);
        img_device = NULL;
    }

    if (tmp_img_device)
    {
        hipFree(tmp_img_device);
        tmp_img_device = NULL;
    }

    hipGraphicsUnregisterResource(cuda_pbo_resource);

    glDeleteBuffers(1, &pbo);
    glDeleteTextures(1, &texture_id);
    glDeleteProgramsARB(1, &shader);

    HANDLE_ERROR(hipFree(particles_d));
    HANDLE_ERROR(hipFree(env_d));
    HANDLE_ERROR(hipFree(occupied_d));
}

int main(int argc, char *argv[])
{
#if defined(__linux__)
    setenv("DISPLAY", ":0", 0);
#endif
    findCudaDevice(argc, (const char **)argv);
    initGL(&argc, argv);
    initCuda();
    initGLResources();
    glutCloseFunc(cleanup);
    glutMainLoop();
    return 0;
}
